
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <string.h>
#include <time.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#include <limits.h>
#include <omp.h>
#include <immintrin.h>
#include <assert.h>
#include <hiprand/hiprand_kernel.h>

#define HEADER_LINES 6
#define NUM_CITIES 13510
#define NUM_DIMS 2

// CUR_THREAD_NUM should be >= num_runs's next power of 2
#define CUR_THREAD_NUM 256
// The total amount of shared memory per block
#define MAX_SHARED_MEM_SIZE_PER_BLOCK 49152

#define msg(format, ...) do { fprintf(stderr, format, ##__VA_ARGS__); } while (0)
#define err(format, ...) do { fprintf(stderr, format, ##__VA_ARGS__); exit(1); } while (0)

#define malloc2D(name, xDim, yDim, type) do {             \
	name = (type **)malloc(xDim * sizeof(type *));          \
	assert(name != NULL);                                   \
	name[0] = (type *)malloc(xDim * yDim * sizeof(type));   \
	assert(name[0] != NULL);                                \
	size_t i;                                               \
	for (i = 1; i < xDim; i++)                              \
		name[i] = name[i-1] + yDim;                           \
} while (0)

int num_runs = 1;

int** city;     // city[id][0] = x of city id, city[id][1] = y of city id,
int** city_dis; // distance between city, city_dis[id1][id2] = distance(id1,id2)
int city_count;
int city_count_int_size;
int city_count_bool_size;

__attribute__((always_inline)) inline void checkCuda(hipError_t e) {
	if (e != hipSuccess) {
		err("CUDA Error:%s\n", hipGetErrorString(e));
	}
}

__attribute__((always_inline)) inline void checkLastCudaError() {
	checkCuda(hipGetLastError());
}

__attribute__((always_inline)) static inline int powerOfTwo(int n) {
	n--;

	n = n >>  1 | n;
	n = n >>  2 | n;
	n = n >>  4 | n;
	n = n >>  8 | n;
	n = n >> 16 | n;
	// n = n >> 32 | n;    //  For 64-bit ints

	return n;
}

__attribute__((always_inline)) inline int distance(int a, int b)
{
	int x_dis = city[a][0] - city[b][0];
	int y_dis = city[a][1] - city[b][1];
	return round(sqrt(x_dis * x_dis + y_dis * y_dis));
}

__attribute__((always_inline)) __device__ inline int route_distance(int *city, int *city_dis, int city_count)
{
	int i, tot_dis = 0;
	int lastIdx = city_count - 1;

	for (i = 0; i < lastIdx - 4; i += 4) {
		// this reduction is memory bound
		tot_dis += (city_dis[city[i] * city_count + city[i + 1]] + city_dis[city[i + 1] * city_count + city[i + 2]]
			+ city_dis[city[i + 2] * city_count + city[i + 3]] + city_dis[city[i + 3] * city_count + city[i + 4]]);
	}
	for (; i < lastIdx; i++)
		tot_dis += city_dis[city[i] * city_count + city[i + 1]];

	tot_dis += city_dis[city[0] * city_count + city[lastIdx]];	// dist for tail ~ head
	return tot_dis;
}

__attribute__((always_inline)) __device__ inline
void gen_neighbor(int *a, int *new_a, int i, int j, int *city_dis, int city_count)
{ // swap the order of ith~jth city of array a and return
	int tmp, k;
	for (tmp = 0; tmp < i; tmp++)
		new_a[tmp] = a[tmp];
	for (tmp = j + 1; tmp < city_count; tmp++)
		new_a[tmp] = a[tmp];
	for (k = 0; k <= j - i; k++) {
		new_a[i + k] = a[j - k];
	}
}

__attribute__((always_inline)) __device__ inline
int neighbor_dis(int *a, int p1, int p2, int ori_part, int *city_dis, int city_count)
{
	int p1_left, p2_right, tmp;
	if (p1 > p2) {
		tmp = p1;
		p1 = p2;
		p2 = tmp;
	}
	int lastCity = city_count - 1;
	p1_left = p1 - 1;
	if (p1 == 0)
		p1_left = lastCity;
	p2_right = p2 + 1;
	if (p2 == lastCity)
		p2_right = 0;

	return city_dis[a[p1_left] * city_count + a[p2]] + city_dis[a[p2_right] * city_count + a[p1]] - ori_part;
}

__device__ bool get_neighbor(int *a, int *new_a, int *city_dis, int city_count)
{
	int i, j, j_right;

	int lastIdx = city_count - 1;
	int beforeLastIdx = city_count - 2;
	float ori_base = city_dis[a[0] * city_count + a[lastIdx]];

	for (i = 0; i < 2;) {
		for (j = i + 1; j < city_count; j++) {
			if (j - i < beforeLastIdx) {
				j_right = j + 1;
				if (j == lastIdx) {
					j_right = 0;
				}
				if (neighbor_dis(a, i, j, ori_base + city_dis[a[j] * city_count + a[j_right]], city_dis, city_count) < 0) {
					gen_neighbor(a, new_a, i, j, city_dis, city_count);
					return true;
				}
			}
		}
		i++;
		ori_base = city_dis[a[i] * city_count + a[i - 1]];
	}

	for (i = 2; i < lastIdx; i++) {
		ori_base = city_dis[a[i] * city_count + a[i - 1]];
		for (j = i + 1; j < lastIdx; j++) {
			if (j - i < beforeLastIdx) {
				if (neighbor_dis(a, i, j, ori_base + city_dis[a[j] * city_count + a[j+1]], city_dis, city_count) < 0) {
					gen_neighbor(a, new_a, i, j, city_dis, city_count);
					return true;
				}
			}
		}
		if (neighbor_dis(a, i, j, ori_base + city_dis[a[j]* city_count + a[0]], city_dis, city_count) < 0) {
			gen_neighbor(a, new_a, i, j, city_dis, city_count);
			return true;
		}
	}
	return false;
}

__attribute__((always_inline)) inline __device__
int hill(int *city_dis, int city_count, hiprandState *state)
{
	int ans;
	int start_state[NUM_CITIES];
	int swapped_state[NUM_CITIES];
	bool gened[NUM_CITIES] = {0};
	int gen_count = 0, gen_id;

	while (gen_count < city_count) {
		gen_id = hiprand(state) % city_count;
		while (gened[gen_id])
			if (++gen_id == city_count)
				gen_id = 0;
		gened[gen_id] = 1;
		start_state[gen_count++] = gen_id;
	}
	int *cur_state = &start_state[0];
	int *new_state = &swapped_state[0];

	while (1) {
		ans = route_distance(cur_state, city_dis, city_count);
		if (!get_neighbor(cur_state, new_state, city_dis, city_count)) {
			break;
		}
		// Swap arrays
		int *temp_ptr = cur_state;
		cur_state = new_state;
		new_state = temp_ptr;
	}

	return ans;
}

__global__ static void calculate_city_distances(
	int cities_count,
	int* dev_cities,      // city[id][0] = x of city id, city[id][1] = y of city id,
	int* dev_cities_dis)  // distance between city, city_dis[id1][id2] = distance(id1,id2)
{
	extern __shared__ int sharedMemory[];
	int *city = (int *)sharedMemory;
	int tid = threadIdx.x;
	int gtid = blockIdx.x * blockDim.x + tid;

	for (int i = tid; i < cities_count*NUM_DIMS; i += blockDim.x) {
		city[i] = dev_cities[i];
	}
	__syncthreads();

	if (gtid < cities_count) {
		// city_dis[city_count - 1][i] = city_dis[i][city_count - 1] = distance(i, city_count - 1);
		for (int i = 0; i < gtid; ++i) {
			int base_p1 = NUM_DIMS*i;
			int base_p2 = NUM_DIMS*gtid;
			int x_dis = city[base_p1] - city[base_p2];
			int y_dis = city[base_p1 + 1] - city[base_p2 + 1];
			int dist = round(sqrtf(x_dis * x_dis + y_dis * y_dis));

			dev_cities_dis[cities_count*gtid + i] = dev_cities_dis[cities_count*i + gtid] = dist;
		}
	}
}

__global__ static void do_hill_climbing_shared(
	int num_runs,
	int city_count,				 // Total # of cities
	int first_step,
	int *dev_cities_dis)   // distance between city, city_dis[id1][id2] = distance(id1,id2)
{
	extern __shared__ int sharedMemory[];
	int *distances = (int *)sharedMemory;
	int *city_dist = (int *)sharedMemory + num_runs;

	for (int i = threadIdx.x; i < city_count*city_count; i += blockDim.x) {
		city_dist[i] = dev_cities_dis[i];
	}
	 __syncthreads();

	if (threadIdx.x < num_runs) {
		hiprandState rng_states;
		hiprand_init(clock64(), threadIdx.x, 0, &rng_states);

		int local_distance = hill(city_dist, city_count, &rng_states);
		printf(" %d", local_distance);
		distances[threadIdx.x] = local_distance;
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		distances[num_runs] = 0;  // Store the reduction result
	}
	for (unsigned int s = first_step; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			int dist = 0, sum = distances[threadIdx.x];
			int idx = threadIdx.x + s;
			if (idx < num_runs) {
				dist = distances[idx];
				if (dist < distances[threadIdx.x]) {
					distances[threadIdx.x] = dist;
				}
			}
			if (s == first_step) {
				atomicAdd(&distances[num_runs], dist + sum);
			}
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		printf("\nMin: %d Avg: %f\n", distances[0], (double) distances[num_runs] / num_runs);
	}
}

__global__ static void do_hill_climbing(
	int num_runs,
	int city_count,				 // Total # of cities
	int first_step,
	int *dev_cities_dis)   // distance between city, city_dis[id1][id2] = distance(id1,id2)
{
	extern __shared__ int sharedMemory[];
	int *distances = (int *)sharedMemory;
	int *city_dist = dev_cities_dis;

	if (threadIdx.x < num_runs) {
		hiprandState rng_states;
		hiprand_init(clock64(), threadIdx.x, 0, &rng_states);
		int local_distance = hill(city_dist, city_count, &rng_states);
		printf(" %d", local_distance);
		distances[threadIdx.x] = local_distance;
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		distances[num_runs] = 0;  // Store the reduction result
	}
	for (unsigned int s = first_step; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			int dist = 0, sum = distances[threadIdx.x];
			int idx = threadIdx.x + s;
			if (idx < num_runs) {
				dist = distances[idx];
				if (dist < distances[threadIdx.x]) {
					distances[threadIdx.x] = dist;
				}
			}
			if (s == first_step) {
				atomicAdd(&distances[num_runs], dist + sum);
			}
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		printf("\nMin: %d Avg: %f\n", distances[0], (double) distances[num_runs] / num_runs);
	}
}

int main(int argc, char **argv)
{
	int optFlag;
	char *file = NULL;
	while ((optFlag = getopt(argc, argv, "f:r:h")) != -1) {
		switch (optFlag) {
		case 'f':
			file = optarg;
			break;
		case 'r':
			num_runs = atoi(optarg);
			break;
		case 'h':
			printf("Usage: ./hill [-h] -f <datafile> -r <num_runs>\n");
			printf("Required:\n");
			printf("  -f <datafile>		Input data file.\n");
			printf("  -r <num>		Number of runs (Default: 1).\n");
			printf("Options:\n");
			printf("  -h              Prints this usage info.\n");
			exit(0);
		default:
			printf("Usage: ./hill [-h] -f <datafile> -r <num_runs>\n");
			exit(0);
		}
	}
	if (!file) {
		printf("Usage: ./hill [-h] -f <datafile> -r <num_runs>\n");
		exit(0);
	}

	double start = omp_get_wtime();
	srand(time(NULL));
	char line[200];
	int jmp_counter = HEADER_LINES;
	float x, y;
	malloc2D(city, NUM_CITIES, NUM_DIMS, int);

	FILE *f;
	f = fopen(file, "r");
	printf("\n%s\n---------------------\n", file);
	city_count = 0;
	while (fgets(line, 198, f) != NULL) {
		if (strstr(line, "EOF") || line[0] == '\n')
			break;
		else if (jmp_counter-- <= 0) { // skip data file header
			sscanf(line, "%*d %E %E", &x, &y);
			city[city_count][0] = (int)x, city[city_count++][1] = (int)y;
		}
	}

	malloc2D(city_dis, city_count, city_count, int);

	const unsigned int cities_size = city_count * NUM_DIMS * sizeof(int);
	const unsigned int city_dis_size = city_count * city_count * sizeof(int);
	const unsigned int cities_quotient = city_count / CUR_THREAD_NUM;
	const unsigned int init_num_blocks = cities_quotient + 1;
	const unsigned int init_block_shared_data_size = cities_size;

	int *dev_cities;     // city[id][0] = x of city id, city[id][1] = y of city id,
	int *dev_cities_dis; // distance between city, city_dis[id1][id2] = distance(id1,id2)

	checkCuda(hipMalloc(&dev_cities, cities_size));
	checkCuda(hipMalloc(&dev_cities_dis, city_dis_size));
	checkCuda(hipMemcpy(dev_cities, city[0], cities_size, hipMemcpyHostToDevice));
	checkCuda(hipMemset(dev_cities_dis, 0, city_dis_size));
	calculate_city_distances<<< init_num_blocks, CUR_THREAD_NUM, init_block_shared_data_size >>>(
		city_count, dev_cities, dev_cities_dis);
	hipDeviceSynchronize(); checkLastCudaError();

	double init = omp_get_wtime();

	city_count_int_size = sizeof(int) * city_count;
	city_count_bool_size = sizeof(bool) * city_count;
	const unsigned int first_step = powerOfTwo(num_runs);
	const unsigned int num_runs_int_size = (num_runs + 1) * sizeof(int); // The last one is for reduction
	const unsigned int hill_block_shared_data_size = city_dis_size + num_runs_int_size;

	printf("Final distances:");
	if (hill_block_shared_data_size <= MAX_SHARED_MEM_SIZE_PER_BLOCK) {
		do_hill_climbing_shared<<< 1, CUR_THREAD_NUM, hill_block_shared_data_size >>>(
			num_runs, city_count, first_step, dev_cities_dis);
	} else {
		do_hill_climbing<<< 1, CUR_THREAD_NUM, num_runs_int_size >>>(
			num_runs, city_count, first_step, dev_cities_dis);
	}
	hipDeviceSynchronize(); checkLastCudaError();

	double end = omp_get_wtime();
	printf("Total initialization time: %f\n", (double)(init - start));
	printf("Total execution time: %f\n", (double)(end - start));

	checkCuda(hipFree(dev_cities));
	checkCuda(hipFree(dev_cities_dis));

	free(city[0]);
	free(city);
	free(city_dis[0]);
	free(city_dis);

	return 0;
}
